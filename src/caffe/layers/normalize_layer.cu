#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include <npp.h>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  Dtype* squared_data = squared_.mutable_gpu_data();
  int n = bottom[0]->num();
  int d = bottom[0]->count() / n;
  //caffe_gpu_powx(n*d, bottom_data, Dtype(2), squared_data);
  for (int i=0; i<n; ++i) {
    //caffe_gpu_asum<Dtype>(d, squared_data+i*d, &normsqr);
    //caffe_gpu_scale<Dtype>(d, pow(10,0.5)*pow(normsqr, -0.5), bottom_data+i*d, top_data+i*d);
    caffe_gpu_scale<Dtype>(d, Dtype(1.0), bottom_data+i*d, top_data+i*d);
    caffe_gpu_add_scalar<Dtype>(d, Dtype(1.0), top_data+i*d);
  }

//  for (int i=0; i<n; ++i) {
//    caffe_gpu_scale<Dtype>(d, Dtype(1.0/50.0), bottom_data+i*d, top_data+i*d);
//    caffe_gpu_add_scalar<Dtype>(d, Dtype(1.0/512.0), top_data+i*d);
//  }
}

//template <typename Dtype>
//void NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
//    vector<Blob<Dtype>*>* top) {
//  const Dtype* bottom_data = bottom[0]->gpu_data();
//  Dtype* top_data = (*top)[0]->mutable_gpu_data();
//  Dtype* squared_data = squared_.mutable_gpu_data();
//  Dtype normsqr;
//  int n = bottom[0]->num();
//  int d = bottom[0]->count() / n;
//	Npp32f min, max;
//	int bytes;
//  for (int i=0; i<n; ++i) {
//	nppsMinMaxGetBufferSize_32f(d,&bytes);
//	Npp8u * scratch = nppsMalloc_8u(bytes*4);
//		nppsMinMax_32f((Npp32f*)(bottom_data + i*d), d, &min, &max, scratch);
//    caffe_gpu_scale<Dtype>(d, Dtype(1.0/50.0), bottom_data+i*d, top_data+i*d);
//    caffe_gpu_add_scalar<Dtype>(d, Dtype(1.0/512.0), top_data+i*d);
//		LOG(INFO) << "Min: " << (float)min << " Max: " << (float)max;
//	nppsFree(scratch);
//  }
//
////  for (int i=0; i<n; ++i) {
////    caffe_gpu_scale<Dtype>(d, Dtype(1.0/50.0), bottom_data+i*d, top_data+i*d);
////    caffe_gpu_add_scalar<Dtype>(d, Dtype(1.0/512.0), top_data+i*d);
////  }
////  caffe_gpu_powx(n*d, bottom_data, Dtype(2), squared_data);
////  for (int i=0; i<n; ++i) {
////    caffe_gpu_asum<Dtype>(d, squared_data+i*d, &normsqr);
////    caffe_gpu_scale<Dtype>(d, pow(normsqr, -0.5), bottom_data+i*d, top_data+i*d);
////    caffe_gpu_add_scalar<Dtype>(d, 1, top_data+i*d);
////  }
//}

template <typename Dtype>
void NormalizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  //const Dtype* top_data = top[0]->gpu_data();
  //const Dtype* bottom_data = (*bottom)[0]->gpu_data();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  //int n = top[0]->num();
  //int d = top[0]->count() / n;
  //Dtype a;
  //for (int i=0; i<n; ++i) {
    //caffe_gpu_dot(d, top_data+i*d, top_diff+i*d, &a);
    //caffe_gpu_scale(d, Dtype(1.0), top_data+i*d, bottom_diff+i*d);
    //caffe_gpu_sub(d, top_diff+i*d, bottom_diff+i*d, bottom_diff+i*d);
    //caffe_gpu_dot(d, bottom_data+i*d, bottom_data+i*d, &a);
    //caffe_gpu_scale(d, Dtype(pow(a, -0.5)), bottom_diff+i*d, bottom_diff+i*d);
  //}
  caffe_copy(top[0]->count(), top_diff, bottom_diff);
}

INSTANTIATE_CLASS(NormalizeLayer);


}  // namespace caffe
